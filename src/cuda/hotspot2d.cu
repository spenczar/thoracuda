#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>

#include <cub/device/device_scan.cuh>

#include "cuda_macros.h"
#include "hotspot2d.h"
#include "pair.h"

hipError_t copy_clusters_to_host(struct Cluster **clusters_d, int *n_clusters_d, struct Cluster **clusters_h,
                                  int *n_clusters_h) {
  hipError_t err;
  CUDA_OR_FAIL(hipMemcpy(n_clusters_h, n_clusters_d, sizeof(int), hipMemcpyDeviceToHost));
  if (*n_clusters_h > 0) {
    *clusters_h = (struct Cluster *)malloc(*n_clusters_h * sizeof(struct Cluster));
    if (*clusters_h == NULL) {
      err = hipErrorOutOfMemory;
      goto fail;
    }
    CUDA_OR_FAIL(hipMemcpy(*clusters_h, *clusters_d, *n_clusters_h * sizeof(struct Cluster), hipMemcpyDeviceToHost));
    for (int i = 0; i < *n_clusters_h; i++) {
      (*clusters_h)[i].point_ids = (int *)malloc((*clusters_h)[i].n * sizeof(int));
      CUDA_OR_FAIL(hipMemcpy((*clusters_h)[i].point_ids, (*clusters_d)[i].point_ids, (*clusters_h)[i].n * sizeof(int),
                              hipMemcpyDeviceToHost));
    }
  }
fail:
  if (err != hipSuccess) {
    if (*clusters_h) {
      for (int i = 0; i < *n_clusters_h; i++) {
        if ((*clusters_h)[i].point_ids) {
          free((*clusters_h)[i].point_ids);
        }
      }
      free(*clusters_h);
      *clusters_h = NULL;
    }
  }
  return err;
}

hipError_t hotspot2d_parallel(const struct XYPair *xys, const float *ts, const int *ids, int n,
                               struct Hotspot2DParams params, struct Cluster **clusters, int *n_clusters) {
  hipError_t err;
  struct XYPair *xys_d;
  float *ts_d;
  int *ids_d;
  struct Cluster **clusters_d;
  int *n_clusters_d;

  CUDA_OR_FAIL(hipMalloc((void **)&xys_d, n * sizeof(struct XYPair)));
  CUDA_OR_FAIL(hipMalloc((void **)&ts_d, n * sizeof(float)));
  CUDA_OR_FAIL(hipMalloc((void **)&ids_d, n * sizeof(int)));
  CUDA_OR_FAIL(hipMalloc((void **)&clusters_d, sizeof(struct Cluster *)));
  CUDA_OR_FAIL(hipMalloc((void **)&n_clusters_d, sizeof(int)));

  CUDA_OR_FAIL(hipMemcpy(xys_d, xys, n * sizeof(struct XYPair), hipMemcpyHostToDevice));
  CUDA_OR_FAIL(hipMemcpy(ts_d, ts, n * sizeof(float), hipMemcpyHostToDevice));
  CUDA_OR_FAIL(hipMemcpy(ids_d, ids, n * sizeof(int), hipMemcpyHostToDevice));

  err = _hotspot2d_parallel_on_device(xys_d, ts_d, ids_d, n, params, clusters_d, n_clusters_d);
  if (err != hipSuccess) {
    goto fail;
  }

  err = copy_clusters_to_host(clusters_d, n_clusters_d, clusters, n_clusters);
  if (err != hipSuccess) {
    goto fail;
  }

fail:
  if (xys_d) {
    hipFree(xys_d);
  }
  if (ts_d) {
    hipFree(ts_d);
  }
  if (ids_d) {
    hipFree(ids_d);
  }
  if (clusters_d) {
    hipFree(clusters_d);
  }
  if (n_clusters_d) {
    hipFree(n_clusters_d);
  }
  return err;
}

hipError_t _hotspot2d_parallel_on_device(const struct XYPair *xys_d, const float *ts_d, const int *ids_d, int n,
                                          struct Hotspot2DParams params, struct Cluster **clusters_d,
                                          int *n_clusters_d) {
  hipError_t err;

  // First, find the "exposure boundaries:" the indexes when t values
  // change in ts_d.
  int *boundaries_d = NULL;
  int n_boundaries_d;
  CUDA_OR_FAIL(find_exposure_boundaries(ts_d, n, &boundaries_d, &n_boundaries_d));

fail:
  if (boundaries_d) {
    hipFree(boundaries_d);
  }
  return hipSuccess;
}

hipError_t find_exposure_boundaries(const float *ts_d, int n, int **boundaries_d, int *n_boundaries_d) {
  // Find all the indexes when timestamp changes. This proceeds in three steps:
  // 1. Map ts_d to 1 if ts_d[i] != ts_d[i-1], 0 otherwise.
  // 2. Inclusive prefix-sum the 1s and 0s.
  // 3. Final value in the prefix sum list is the n_boundaries
  // 4. Allocate boundaries_d
  // 5. Extract the boundary points by compacting using the prefix sums.

  hipError_t err;
  int n_threads = 256;
  int n_blocks = (n + n_threads - 1) / n_threads;
  void *temp_storage_d = NULL;
  size_t temp_storage_bytes = 0;

  // Part 1: Mark changes
  int *changes_d;
  CUDA_OR_FAIL(hipMalloc((void **)&changes_d, n * sizeof(int)));

  mark_changes_kernel<<<n_blocks, n_threads>>>(ts_d, changes_d, n);
  CUDA_CHECK_ERROR();

  // Part 2: Prefix sum
  int *summed_changes_d;
  CUDA_OR_FAIL(hipMalloc((void **)&summed_changes_d, n * sizeof(int)));
  // Determine storage requirements for a CUB prefix scan over changes_d
  CUDA_OR_FAIL(hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, changes_d, summed_changes_d, n));
  CUDA_OR_FAIL(hipMalloc(&temp_storage_d, temp_storage_bytes));
  CUDA_OR_FAIL(hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, changes_d, summed_changes_d, n));
  hipFree(temp_storage_d);
  temp_storage_d = NULL;

  // Part 3: Determine n_boundaries
  int n_boundaries_h;

  CUDA_OR_FAIL(hipMemcpy(&n_boundaries_h, summed_changes_d + n - 1, sizeof(int), hipMemcpyDeviceToHost));

  assert(n_boundaries_h >= 0);

  CUDA_OR_FAIL(hipMemcpy(n_boundaries_d, &n_boundaries_h, sizeof(int), hipMemcpyHostToDevice));

  // Part 4: Allocate boundaries_d
  int *boundaries;
  CUDA_OR_FAIL(hipMalloc((void **)&boundaries, n_boundaries_h * sizeof(int)));

  // Part 5: Extract boundary points
  gather_change_indexes_kernel<<<n_blocks, n_threads>>>(summed_changes_d, boundaries, n);
  CUDA_CHECK_ERROR();

  *boundaries_d = boundaries;
  err = hipSuccess;

fail:
  if (changes_d) {
    hipFree(changes_d);
  }
  if (summed_changes_d) {
    hipFree(summed_changes_d);
  }
  if (temp_storage_d) {
    hipFree(temp_storage_d);
  }
  return err;
}

// 1.0 1.0  2.0 3.0 3.0 4.0
// 0   0    1   1   0   1
// 0   0    1   2   2   3
__global__ void mark_changes_kernel(const float *in, int *out, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i == 0) {
    out[0] = 0;
    i += blockDim.x * gridDim.x;
  }
  for (; i < n; i += blockDim.x * gridDim.x) {
    out[i] = in[i] != in[i - 1];
  }
}

__global__ void gather_change_indexes_kernel(const int *summed_changes, int *out, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i == 0) {
    out[0] = 0;
    i += blockDim.x * gridDim.x;
  }
  for (; i < n; i += blockDim.x * gridDim.x) {
    if (summed_changes[i] != summed_changes[i - 1]) {
      out[summed_changes[i]] = i;
    }
  }
}