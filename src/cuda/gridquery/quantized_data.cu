#include "hip/hip_runtime.h"
#include <stdexcept>

#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <hip/hip_runtime.h>
#include <cuda/std/tuple>

#include "gridquery/quantized_data.cuh"

#include "pair.h"
#include "pairminmax.h"
#include "rangequery/data_handle.cuh"
#include "rangequery/row.cuh"
#include "cuda_macros.h"

using thoracuda::rangequery::DataHandle;

namespace thoracuda {
namespace gridquery {

  QuantizedData::QuantizedData() {
    this->quantized = nullptr;
    this->n = 0;
  }

  QuantizedData::QuantizedData(DataHandle &dh, int n_cells) {
    hipError_t err;
    int n_threads = 256;
    int n_blocks = (dh.n + n_threads - 1) / n_threads;
    
    // First, establish bounds of the data.
    struct XYBounds *bounds = nullptr;
    struct XYPair *pairs = nullptr;
    struct XYBounds *host_bounds = new struct XYBounds;
    
    CUDA_OR_FAIL(hipMalloc(&pairs, dh.n * sizeof(struct XYPair)));
    copy_rows_to_xypairs<<<n_blocks, n_threads>>>(dh.rows, dh.n, pairs);
    CUDA_OR_FAIL(hipGetLastError());

    // Now, find the min and max of the data.
    CUDA_OR_FAIL(xy_bounds_parallel(pairs, dh.n, host_bounds));

    // Now, quantize the data.
    CUDA_OR_FAIL(hipMalloc(&this->quantized, dh.n * sizeof(int2)));
    quantize_data<<<n_blocks, n_threads>>>(pairs, dh.n, *host_bounds, n_cells, this->quantized);
    hipDeviceSynchronize();
    CUDA_OR_FAIL(hipGetLastError());
    this->n = dh.n;

  fail:
    if (bounds != nullptr) {
      delete bounds;
    }
    if (pairs != nullptr) {
      hipFree(pairs);
    }
    if (host_bounds != nullptr) {
      delete host_bounds;
    }
    if (err != hipSuccess) {
      throw std::runtime_error(hipGetErrorString(err));
    }
  }

  QuantizedData::QuantizedData(DataHandle &dh, int n_cells, struct XYBounds bounds) {
    hipError_t err;
    int n_threads = 256;
    int n_blocks = (dh.n + n_threads - 1) / n_threads;
    
    CUDA_OR_THROW(hipMalloc(&this->quantized, dh.n * sizeof(int2)));

    // Copy the rows to a new array of XYPairs.
    struct XYPair *pairs = nullptr;
    CUDA_OR_FAIL(hipMalloc(&pairs, dh.n * sizeof(struct XYPair)));
    copy_rows_to_xypairs<<<n_blocks, n_threads>>>(dh.rows, dh.n, pairs);
    CUDA_OR_THROW(hipGetLastError());
    
    // Quantize the data.
    quantize_data<<<n_blocks, n_threads>>>(pairs, dh.n, bounds, n_cells, this->quantized);
    
    hipDeviceSynchronize();
    CUDA_OR_THROW(hipGetLastError());
    
    this->n = dh.n;

  fail:
    if (pairs != nullptr) {
      hipFree(pairs);
    }
    if (err != hipSuccess) {
      throw std::runtime_error(hipGetErrorString(err));
    }
  }

  QuantizedData::~QuantizedData() {
    if (this->quantized != nullptr) {
      hipFree(this->quantized);
    }
  }

  QuantizedData::QuantizedData(QuantizedData &&other) {
    this->quantized = other.quantized;
    this->n = other.n;
    other.quantized = nullptr;
    other.n = 0;
  }

  QuantizedData &QuantizedData::operator=(QuantizedData &&other) {
    if (this->quantized != nullptr) {
      hipFree(this->quantized);
    }
    this->quantized = other.quantized;
    this->n = other.n;
    other.quantized = nullptr;
    other.n = 0;
    return *this;
  }

  std::vector<int2> QuantizedData::to_host_vector() const {
    hipError_t err;
    std::vector<int2> result;
    result.reserve(this->n);
    int *host_quantized = new int[this->n];
    
    CUDA_OR_THROW(hipMemcpy(host_quantized, this->quantized, this->n * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < this->n; i++) {
      int x = host_quantized[i] >> 16;
      int y = host_quantized[i] & 0xFFFF;
      result.push_back(make_int2(x, y));
    }
    delete[] host_quantized;
    return result;
  }
  
  __global__ void copy_rows_to_xypairs(thoracuda::rangequery::Row *rows, int n, struct XYPair *pairs) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (; i < n; i += blockDim.x * gridDim.x) {
      pairs[i].x = rows[i].x;
      pairs[i].y = rows[i].y;
    }
  }

  __global__ void quantize_data(struct XYPair *pairs, int n, struct XYBounds bounds, int n_cells, int *quantized) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
      return;
    }
    for (; i < n; i += blockDim.x * gridDim.x) {
      float x = pairs[i].x;
      float y = pairs[i].y;
      int cell_x = (x - bounds.xmin) / (bounds.xmax - bounds.xmin) * n_cells;
      int cell_y = (y - bounds.ymin) / (bounds.ymax - bounds.ymin) * n_cells;
      quantized[i] = cell_x << 16 | cell_y;
    } 
  }
}
}